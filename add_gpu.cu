#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdint>
#include <Python.h>
#include "dlpack.h"

__global__
void kernel_add(float* a, size_t length, float value) {
    size_t idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < length) {
        a[idx] += idx + value;
    }
}

void print_info(const DLTensor* tensor)
{
    printf("context : %d %d\n", tensor->ctx.device_type, tensor->ctx.device_id);
    printf("dtype : %d %d %d\n", tensor->dtype.code, tensor->dtype.bits, tensor->dtype.lanes);
    printf("ndim : %d\n", tensor->ndim);
    printf("shape : ");
    size_t len = 1;
    for (int i=0; i<tensor->ndim; ++i) {
        len *= tensor->shape[i];
        printf("%ld ", tensor->shape[i]);
    }
    printf("\n");
    printf("strides : ");
    for (int i=0; i<tensor->ndim; ++i) {
        printf("%ld ", tensor->strides[i]);
    }
    printf("\n");
    printf("byte_offset : %lu\n", tensor->byte_offset);
}

void cadd(void* obj, float value)
{
//   printf("%p %d\n", obj, value);
    DLTensor* tensor = (DLTensor*)obj;
    // print_info(tensor);
    size_t len = 1;
    for (int i=0; i<tensor->ndim; ++i) {
        len *= tensor->shape[i];
    }
    float* data = (float*)tensor->data;
    kernel_add<<<(len+127)/128, 128>>>(data, len, value);
}
